#include "hip/hip_runtime.h"
﻿#define _CRT_SECURE_NO_WARNINGS
#include "cstdio"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <string>
#include <cassert>
#define BLUR_SIZE 3
#define R 0
#define G 1
#define B 2
#define A 3

// TODO:
// Zmienic sciezke do pliku neon.png
// zbudowac wersje
const char * PATH = "C:/Users/krzys/source/repos/CudaRuntime5/x64/Release/neon.png";


struct Pixel
{
    unsigned char r, g, b, a;
};

__global__ void ConvertImageToGrayGpu(unsigned char* imageRGBA)
{
    
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t idx = y * blockDim.x * gridDim.x + x;

    Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
    unsigned char pixelValue = (unsigned char)
        (ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
    ptrPixel->r = pixelValue;
    ptrPixel->g = pixelValue;
    ptrPixel->b = pixelValue;
    ptrPixel->a = 255;
}

__global__ void CovertImageToSepiaGpu(unsigned char* imageRGBA)
{
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t idx = y * blockDim.x * gridDim.x + x;

    Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];

    int nowy_r = ptrPixel->r * 0.392f + ptrPixel->g * 0.769f + ptrPixel->b * 0.189f;
    int nowy_g = ptrPixel->r * 0.349f + ptrPixel->g * 0.686f + ptrPixel->b * 0.168f;
    int nowy_b = ptrPixel->r * 0.272f + ptrPixel->g * 0.534f + ptrPixel->b * 0.131f;

    nowy_r > 255 ? nowy_r = 255 : nowy_r = nowy_r;
    nowy_g > 255 ? nowy_g = 255 : nowy_g = nowy_g;
    nowy_b > 255 ? nowy_b = 255 : nowy_b = nowy_b;

    ptrPixel->r = nowy_r;
    ptrPixel->g = nowy_g;
    ptrPixel->b = nowy_b;
    ptrPixel->a = 255; 
}

__global__ void CovertImageInverseGpu(unsigned char* imageRGBA)
{
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t idx = y * blockDim.x * gridDim.x + x;

    Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
    ptrPixel->r = 255 - ptrPixel->r;
    ptrPixel->g = 255 - ptrPixel->g;
    ptrPixel->b = 255 - ptrPixel->b;
    ptrPixel->a = 255;
}

__global__ void CovertImageToBlackAndWhiteGpu(unsigned char* imageRGBA)
{
    int b_or_w_pixel;

    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t idx = y * blockDim.x * gridDim.x + x;

    Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];    

    int avg =(ptrPixel->r + ptrPixel->g + ptrPixel->b) / 3;

   avg >= 100 ? b_or_w_pixel = 255 : b_or_w_pixel = 0;
    
   ptrPixel->r = b_or_w_pixel;
   ptrPixel->g = b_or_w_pixel;
   ptrPixel->b = b_or_w_pixel;
   ptrPixel->a = 255;
}

__global__ void ConvertImageToBlurGpu(unsigned char* in, unsigned char* out, int width, int height, int channels, int channel, int cA) {

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        int pixelValue = 0;
        int pixels = 0;
        if (cA)
            out[row * width * channels + col * channels + A] = in[row * width * channels + col * channels + A];
        for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow) {
            for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol) {
                int curRow = row + blurRow;
                int curCol = col + blurCol;
                if (curRow > -1 && curRow < height && curCol > -1 && curCol < width) {
                    pixelValue += in[curRow * width * channels + curCol * channels + channel];
                    pixels++;
                }
            }
        }
        out[row * width * channels + col * channels + channel] = (unsigned char)(pixelValue / pixels);
    }
}

void invertImageWrapper(unsigned char* imageData, int width, int height) {
       
    std::cout << "Kopiowanie do GPU.....";
    unsigned char* ptrImageDataGpu = nullptr;
    hipMalloc(&ptrImageDataGpu, width * height * 4);
    hipMemcpy(ptrImageDataGpu, imageData, width * height * 4, hipMemcpyHostToDevice);
    std::cout << "ZAKONCZONO" << std::endl;

    std::cout << "KERNEL PRACUJE....";
    dim3 blockSize(16, 16);
    dim3 gridSize(width / blockSize.x, height / blockSize.y);
    CovertImageInverseGpu << <gridSize, blockSize >> > (ptrImageDataGpu);
    auto err = hipGetLastError();
    std::cout << "ZAKONCZONO" << std::endl;

    std::cout << "Kopiowanie z GPU...";
    hipMemcpy(imageData, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost);
    std::cout << "ZAKONCZONO" << std::endl;

    hipFree(ptrImageDataGpu);
}

void grayImageWrapper(unsigned char* imageData, int width, int height) {
       
    std::cout << "Kopiowanie do GPU.....";
    unsigned char* ptrImageDataGpu = nullptr;
    hipMalloc(&ptrImageDataGpu, width * height * 4);
    hipMemcpy(ptrImageDataGpu, imageData, width * height * 4, hipMemcpyHostToDevice);
    std::cout << "ZAKONCZONO" << std::endl;

    std::cout << "KERNEL PRACUJE....";
    dim3 blockSize(16, 16);
    dim3 gridSize(width / blockSize.x, height / blockSize.y);
    ConvertImageToGrayGpu << <gridSize, blockSize >> > (ptrImageDataGpu);
    auto err = hipGetLastError();
    std::cout << "ZAKONCZONO" << std::endl;


    std::cout << "Kopiowanie z GPU...";
    hipMemcpy(imageData, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost);
    std::cout << "ZAKONCZONO" << std::endl;

    hipFree(ptrImageDataGpu);
}

void sepiaImageWrapper(unsigned char* imageData, int width, int height) {

    std::cout << "Kopiowanie do GPU.....";
    unsigned char* ptrImageDataGpu = nullptr;
    hipMalloc(&ptrImageDataGpu, width * height * 4);
    hipMemcpy(ptrImageDataGpu, imageData, width * height * 4, hipMemcpyHostToDevice);
    std::cout << "ZAKONCZONO" << std::endl;

    std::cout << "KERNEL PRACUJE....";
    dim3 blockSize(16, 16);
    dim3 gridSize(width / blockSize.x, height / blockSize.y);
    CovertImageToSepiaGpu << <gridSize, blockSize >> > (ptrImageDataGpu);
    auto err = hipGetLastError();
    std::cout << "ZAKONCZONO" << std::endl;

    std::cout << "Kopiowanie z GPU...";
    hipMemcpy(imageData, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost);
    std::cout << "ZAKONCZONO" << std::endl;

    hipFree(ptrImageDataGpu);
}

void blackAndWhiteImageWrapper(unsigned char* imageData, int width, int height) {

    std::cout << "Kopiowanie do GPU.....";
    unsigned char* ptrImageDataGpu = nullptr;
    hipMalloc(&ptrImageDataGpu, width * height * 4);
    hipMemcpy(ptrImageDataGpu, imageData, width * height * 4, hipMemcpyHostToDevice);
    std::cout << "ZAKONCZONO" << std::endl;

    std::cout << "KERNEL PRACUJE....";
    dim3 blockSize(16, 16);
    dim3 gridSize(width / blockSize.x, height / blockSize.y);
    CovertImageToBlackAndWhiteGpu << <gridSize, blockSize >> > (ptrImageDataGpu);
    auto err = hipGetLastError();
    std::cout << "ZAKONCZONO" << std::endl;

    std::cout << "Kopiowanie z GPU...";
    hipMemcpy(imageData, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost);
    std::cout << "ZAKONCZONO" << std::endl;

    hipFree(ptrImageDataGpu);
}


void blurImageWrapper(unsigned char* imageData,unsigned char* output, int width, int height) {

    int n = 4; // liczba kanalow
    unsigned char* Dev_Input_Image = NULL;
    unsigned char* Dev_Output_Image = NULL;

    std::cout << "Kopiowanie do GPU.....";
    hipMalloc((void**)&Dev_Input_Image, sizeof(unsigned char) * height * width * n);
    hipMalloc((void**)&Dev_Output_Image, sizeof(unsigned char) * height * width * n);    
    hipMemcpy(Dev_Input_Image, imageData, sizeof(unsigned char) * height * width * n, hipMemcpyHostToDevice);
    std::cout << "ZAKONCZONO" << std::endl;

    std::cout << "KERNEL PRACUJE....";
    dim3 blockSize(16, 16, 1);
    dim3 gridSize(width / blockSize.x, height / blockSize.y, 1);

    // Blur na odpowiednich kanalach R G B 
    ConvertImageToBlurGpu << <gridSize, blockSize >> > (Dev_Input_Image, Dev_Output_Image, width, height, n, R, 0);
    ConvertImageToBlurGpu << <gridSize, blockSize >> > (Dev_Input_Image, Dev_Output_Image, width, height, n, G, 0);
    ConvertImageToBlurGpu << <gridSize, blockSize >> > (Dev_Input_Image, Dev_Output_Image, width, height, n, B, 1);

    hipDeviceSynchronize();
    std::cout << "ZAKONCZONO" << std::endl;

    std::cout << "Kopiowanie z GPU...";
    hipMemcpy(imageData, Dev_Output_Image, sizeof(unsigned char) * height * width * n, hipMemcpyDeviceToHost);
    std::cout << "ZAKONCZONO" << std::endl;

    hipFree(Dev_Input_Image);
    hipFree(Dev_Output_Image);
}



int main(int argc, char** argv)
{
   int width, height, componentCount;

    std::cout << "Wczytywanie pliku....";
    system("pause");
    unsigned char* imageData = stbi_load(PATH, &width, &height, &componentCount, 4);
    if (!imageData)
    {
        std::cout << std::endl << "Nie wczytano pliku! (Prawdopobona konieczna edycja sciezki i ponowne zbudowanie projektu)"  << std::endl;
        system("pause");
        return -1;
    }
    std::cout << "ZAKONCZONO" << std::endl;
   

    // Weryfikacja rozmiaru zdjecia (wielokrotnosc liczby 16)
    if (width % 16 || height % 16)
    {
        std::cout << "Wymary obrazka nie sa wielokrotnoscia liczby 16!" << std::endl;
        return -1;
    }
    
    // FILTRY
    
    // INVERT
    std::cout << "--------------INVERT IMAGE--------------" << std::endl;
    invertImageWrapper(imageData, width, height);
    // Zapisywanie pliku
    std::cout << "Zapisywanie pliku...";
    stbi_write_png("inverted_neon.png", width, height, 4, imageData, 4 * width);
    std::cout << "ZAKONCZONO :)" << std::endl;
    std::cout << "--------------------------------" << std::endl;
    // Zwolnienie pamięci    
    stbi_image_free(imageData);
    system("pause");
       
    // GRAY
    std::cout << "--------------GRAY IMAGE--------------" << std::endl;
    imageData = stbi_load(PATH, &width, &height, &componentCount, 4);
    grayImageWrapper(imageData, width, height);
    // Zapisywanie pliku
    std::cout << "Zapisywanie pliku...";
    stbi_write_png("gray_neon.png", width, height, 4, imageData, 4 * width);
    std::cout << "ZAKONCZONO :)" << std::endl;
    std::cout << "--------------------------------" << std::endl;
    // Zwolnienie pamięci 
    stbi_image_free(imageData);
    system("pause");

    // SEPIA
    std::cout << "--------------SEPIA IMAGE--------------" << std::endl;
    imageData = stbi_load(PATH, &width, &height, &componentCount, 4);
    sepiaImageWrapper(imageData, width, height);
    // Zapisywanie pliku
    std::cout << "Zapisywanie pliku...";
    stbi_write_png("sepia_neon.png", width, height, 4, imageData, 4 * width);
    std::cout << "ZAKONCZONO :)" << std::endl;
    std::cout << "--------------------------------" << std::endl;
    // Zwolnienie pamięci 
    stbi_image_free(imageData);
    system("pause");

    // BLACK AND WHITE
    std::cout << "--------------B&W IMAGE--------------" << std::endl;
    imageData = stbi_load(PATH, &width, &height, &componentCount, 4);
    blackAndWhiteImageWrapper(imageData, width, height);
    // Zapisywanie pliku
    std::cout << "Zapisywanie pliku...";
    stbi_write_png("bw_neon.png", width, height, 4, imageData, 4 * width);
    std::cout << "ZAKONCZONO :)" << std::endl;
    std::cout << "--------------------------------" << std::endl;
    // Zwolnienie pamięci 
    stbi_image_free(imageData);
    system("pause");

    // BLUR
    std::cout << "--------------BLURED IMAGE--------------" << std::endl;
    imageData = stbi_load(PATH, &width, &height, &componentCount, 4);      
    unsigned char* output = (unsigned char*)malloc(width * height * 4 * sizeof(unsigned char));
    blurImageWrapper(imageData, output, width, height);       
    // Zapisywanie pliku
    std::cout << "Zapisywanie pliku...";
    stbi_write_png("blur_neon.png", width, height, 4, imageData, 4 * width);
    std::cout << "ZAKONCZONO :)" << std::endl;
    std::cout << "--------------------------------" << std::endl;
    // Zwolnienie pamięci 
    stbi_image_free(imageData);
    system("pause");


}
